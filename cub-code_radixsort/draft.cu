//#include "../../cub-1.8.0/hipcub/hipcub.hpp"   // or equivalently <cub/device/device_histogram.cuh>
#include "hipcub/hipcub.hpp"
#include "helper.cu.h"

template<class Z>
bool validateZ(Z* A, uint32_t sizeAB) {
    for(uint32_t i = 1; i < sizeAB; i++)
      if (A[i-1] > A[i]){
        printf("INVALID RESULT for i:%d, (A[i-1]=%d > A[i]=%d)\n", i, A[i-1], A[i]);
        return false;
      }
    return true;
}

bool validateSegmentedZ(uint32_t* A, uint32_t* offsets, uint32_t num_segments) {
    for(uint32_t seg = 0; seg < num_segments; seg++) {
        uint32_t start = offsets[seg];
        uint32_t end = offsets[seg + 1];
        for(uint32_t i = start + 1; i < end; i++) {
            if (A[i-1] > A[i]){
                printf("INVALID RESULT in segment %d for i:%d, (A[i-1]=%d > A[i]=%d)\n", 
                       seg, i, A[i-1], A[i]);
                return false;
            }
        }
    }
    return true;
}

void randomInitNat(uint32_t* data, const uint32_t size, const uint32_t H) {
    for (int i = 0; i < size; ++i) {
        unsigned long int r = rand();
        data[i] = r % H;
    }
}

void initSegmentOffsets(uint32_t* offsets, uint32_t num_segments, uint64_t N) {
    uint64_t seg_size = N / num_segments;
    for(uint32_t i = 0; i < num_segments; i++) {
        offsets[i] = i * seg_size;
    }
    offsets[num_segments] = N;  // last offset is the total size
}

double sortRedByKeySegmentedCUB( uint32_t* data_keys_in
                               , uint32_t* data_keys_out
                               , uint32_t* d_offsets
                               , const uint64_t N
                               , const uint32_t num_segments
) {
    int beg_bit = 0;
    int end_bit = 32;

    void * tmp_sort_mem = NULL;
    size_t tmp_sort_len = 0;

    { // sort prelude
        hipcub::DeviceSegmentedRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                               , data_keys_in, data_keys_out
                                               , N, num_segments, d_offsets, d_offsets + 1
                                               , beg_bit, end_bit
                                               );
        hipMalloc(&tmp_sort_mem, tmp_sort_len);
    }
    cudaCheckError();

    { // one dry run
        hipcub::DeviceSegmentedRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                               , data_keys_in, data_keys_out
                                               , N, num_segments, d_offsets, d_offsets + 1
                                               , beg_bit, end_bit
                                               );
        hipDeviceSynchronize();
    }
    cudaCheckError();

    // timing
    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    for(int k=0; k<GPU_RUNS; k++) {
        hipcub::DeviceSegmentedRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                               , data_keys_in, data_keys_out
                                               , N, num_segments, d_offsets, d_offsets + 1
                                               , beg_bit, end_bit
                                               );
    }
    hipDeviceSynchronize();
    cudaCheckError();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);

    hipFree(tmp_sort_mem);

    return elapsed;
}


int main (int argc, char * argv[]) {
    if (argc != 3) {
        printf("Usage: %s <size-of-array> <num-segments>\n", argv[0]);
        exit(1);
    }
    const uint64_t N = atoi(argv[1]);
    const uint32_t num_segments = atoi(argv[2]);

    if (N % num_segments != 0) {
        printf("Warning: N should be divisible by num_segments for even segment sizes\n");
    }

    //Allocate and Initialize Host data with random values
    uint32_t* h_keys  = (uint32_t*) malloc(N*sizeof(uint32_t));
    uint32_t* h_keys_res  = (uint32_t*) malloc(N*sizeof(uint32_t));
    uint32_t* h_offsets = (uint32_t*) malloc((num_segments + 1)*sizeof(uint32_t));
    randomInitNat(h_keys, N, N/10);
    initSegmentOffsets(h_offsets, num_segments, N);

    //Allocate and Initialize Device data
    uint32_t* d_keys_in;
    uint32_t* d_keys_out;
    uint32_t* d_offsets;
    cudaSucceeded(hipMalloc((void**) &d_keys_in,  N * sizeof(uint32_t)));
    cudaSucceeded(hipMemcpy(d_keys_in, h_keys, N * sizeof(uint32_t), hipMemcpyHostToDevice));
    cudaSucceeded(hipMalloc((void**) &d_keys_out, N * sizeof(uint32_t)));
    cudaSucceeded(hipMalloc((void**) &d_offsets, (num_segments + 1) * sizeof(uint32_t)));
    cudaSucceeded(hipMemcpy(d_offsets, h_offsets, (num_segments + 1) * sizeof(uint32_t), hipMemcpyHostToDevice));

    double elapsed = sortRedByKeySegmentedCUB( d_keys_in, d_keys_out, d_offsets, N, num_segments );

    hipMemcpy(h_keys_res, d_keys_out, N*sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    cudaCheckError();

    bool success = validateSegmentedZ(h_keys_res, h_offsets, num_segments);

    printf("CUB Segmented Sorting for N=%lu, segments=%u runs in: %.2f us, VALID: %d\n", 
           N, num_segments, elapsed, success);

    // Cleanup and closing
    hipFree(d_keys_in); hipFree(d_keys_out); hipFree(d_offsets);
    free(h_keys); free(h_keys_res); free(h_offsets);

    return success ? 0 : 1;
}